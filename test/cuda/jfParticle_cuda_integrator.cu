
#include <hip/hip_runtime.h>
#include <stdio.h>

// Prototypes
//__global__ void helloWorld(char*);

extern "C" void integrate(float timestep)
{
/**    int i;

    // desired output
    char str[] = "Hello World!";

    // mangle contents of output
    // the null character is left intact for simplicity
    for(i = 0; i < 12; i++)
        str[i] -= i;

    // allocate memory on the device
    char *d_str;
    size_t size = sizeof(str);
    //cudaMalloc((void**)&d_str, size);

    // copy the string to the device
    //cudaMemcpy(d_str, str, size, cudaMemcpyHostToDevice);

    // set the grid and block sizes
    //dim3 dimGrid(2); // one block per word
    //dim3 dimBlock(6); // one thread per character

    // invoke the kernel
    //helloWorld<<< dimGrid, dimBlock >>>(d_str);

    // retrieve the results from the device
    //cudaMemcpy(str, d_str, size, cudaMemcpyDeviceToHost);

    // free up the allocated memory on the device
    //cudaFree(d_str);

    // everyone's favorite part
    printf("%s\n", str);
    **/
}

