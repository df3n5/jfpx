#include "hip/hip_runtime.h"

#include <cuda/hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Device kernel
__global__ void
helloWorld(char* str)
{
    // determine where in the thread grid we are
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // unmangle output
    str[idx] += idx;
}
