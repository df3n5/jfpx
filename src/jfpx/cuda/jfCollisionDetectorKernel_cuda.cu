#include "hip/hip_runtime.h"

#include <jfpx/cuda/jfVector3_cuda.h>
#include <jfpx/cuda/jfContact_cuda.h>
#include <jfpx/cuda/jfCollisionSphere_cuda.h>
#include <jfpx/cuda/jfCudaConstants.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

__device__ float3
multiply(float3 a, float b)
{
    float3 result;
    result.x = a.x * b;
    result.y = a.y * b;
    result.z = a.z * b;
    return result;
}

/**
  * Device function run on GPU which tests
  * whether two spheres are colliding and 
  * if so, generates a contact.
  */
__device__ void
sphereSphereCollision(float3* contactNormal,
                      float4* contactPointPenetration,
                      int* valid,
                      float4 currentSphere,
                      float4 otherSphere
                      )
{
    float3 midline;
    midline.x = currentSphere.x-otherSphere.x;
    midline.y = currentSphere.y-otherSphere.y;
    midline.z = currentSphere.z-otherSphere.z;
    float size = sqrt((midline.x*midline.x) + (midline.y*midline.y) + (midline.z*midline.z));
    float radiiSum = currentSphere.w+otherSphere.w;

    //Don't want us to take it if we have inaccuracy.
    float tolerance = 0.0001;
    if((size > (0.0f+tolerance)) && (size < radiiSum))
    {
        (*valid) = 1;
    }
    else
    {
        (*valid) = 0;
    }

	//Set contact normal
	float invSize = 1.0f/size;
	contactNormal->x = midline.x * invSize;
	contactNormal->y = midline.y * invSize;
	contactNormal->z = midline.z * invSize;

    //Set contact point
    contactPointPenetration->x = (currentSphere.x + (midline.x*0.5));
    contactPointPenetration->y = (currentSphere.y + (midline.y*0.5));
    contactPointPenetration->z = (currentSphere.z + (midline.z*0.5));

    //Set penetration of contact
    contactPointPenetration->w = (radiiSum - size);
}

/**
  * Called at the beginning of checking all of the spheres as this is the only case when the 
  * number of spheres tested is not uniform.
  */
__device__ void
tileSphereSphereCalculationStart(float3* currentContactNormal,
									float4* currentContactPointPenetration,
									int* valid,
									float4 currentSphere,
									float4 sharedSpheres[BLOCK_SIZE],
									unsigned amount)
{
    int i;

    for (i=0; i < amount; i++)
    {
		sphereSphereCollision(&(currentContactNormal[i]),
				&(currentContactPointPenetration[i]),
				&(valid[i]),
				currentSphere,
				sharedSpheres[threadIdx.x+1+i]);
    }
    return;
}

/**
  * Perform collision detection between a tile of spheres
  * in shared memory.
  */
__device__ void
tileSphereSphereCalculation(float3* currentContactNormal,
                            float4* currentContactPointPenetration,
                            int* valid,
                            float4 currentSphere,
                            float4 sharedSpheres[BLOCK_SIZE])
{
    int i;
    for (i=0; i < blockDim.x; i++)
    {
		sphereSphereCollision(&(currentContactNormal[i]),
				&(currentContactPointPenetration[i]),
				&(valid[i]),
				currentSphere,
				sharedSpheres[i]);
    }
    return;
}

/** 
 * Computes the row index in the triangular array.
 * Each row gets shorter and shorter by one each time.
 */
__device__
unsigned computeContactRowIndex(unsigned n_Spheres, unsigned sphere_row)
{
    unsigned rowLength=(n_Spheres-1);
    unsigned resultRow=0;
    for(unsigned i=0;i<sphere_row;i++)
    {
        resultRow+=rowLength;
        --rowLength;
    }
    return resultRow;
}

/**
 * Computes the number of columns in the row of the
 * triangular contacts array
 */
__device__
unsigned computeContactNumCols(unsigned sphere_row)
{
    return ((N_SPHERES-1)-sphere_row);
}

/**
 * Kernel which checks exhaustively for collisions 
 * between the spheres passed in.
 */
__global__ void
calculateContacts(float4* devContactNormal,
                  float4* devContactPointPenetration,
                  int4* devContactValidBodies,
                  float4* devSphere,
				  unsigned numSpheres)
{
    //Define shared memory inside a block
    //Spheres represented by 3 floats for position and 4th for radius
	__shared__ float4 sharedSpheres[BLOCK_SIZE];

    float4* globalSpheres = (float4*) devSphere;

    float4 mySphere;
    int i;
    int tile;
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;

    //Make float3 as we want to conserve registers
    //We will only use as much as we need.
    //Set all to maximum possible size
    float3 currentContactNormal[N_SPHERES-1];
    float4 currentContactPointPenetration[N_SPHERES-1];
    int currentContactValid[N_SPHERES-1];

    unsigned rowIndex = computeContactRowIndex(N_SPHERES, gtid);
	unsigned numCols = computeContactNumCols(gtid);

    //Make float4 as we want to coalesce memory accesses
    float4* globalContactNormal = (float4*) &(devContactNormal[rowIndex]);
    float4* globalContactPointPenetration = (float4*) &(devContactPointPenetration[rowIndex]);
    int4* globalContactValidBodies = (int4*) &(devContactValidBodies[rowIndex]);

    //Get current sphere to test all others against
    mySphere = globalSpheres[gtid];

	i = 0;
	
	unsigned amountContactsFirstBlock = (numCols % BLOCK_SIZE);
	tile = blockIdx.x;
    int idx = tile * blockDim.x + threadIdx.x;
    //Collaboratively load sharedSpheres
    sharedSpheres[threadIdx.x] = globalSpheres[idx];
    __syncthreads();
	//Do the first loop, as it is slightly different, not dealing with square blocks, triangular ones instead
	tileSphereSphereCalculationStart(&(currentContactNormal[i]),
								&(currentContactPointPenetration[i]),
								&(currentContactValid[i]),
								mySphere,
								sharedSpheres,
								amountContactsFirstBlock); //don't skip too far ahead, triangular array first
	__syncthreads();

    //Start at our blockID, this ensures we don't check ones
    //that have been done by other blocks
	for (i = amountContactsFirstBlock, tile = (blockIdx.x+1); i < numCols; i += BLOCK_SIZE, tile++)
    {
        idx = tile * blockDim.x + threadIdx.x;
        //Collaboratively load sharedSpheres
        sharedSpheres[threadIdx.x] = globalSpheres[idx];
        __syncthreads();

        tileSphereSphereCalculation(&(currentContactNormal[i]),
                                    &(currentContactPointPenetration[i]),
                                    &(currentContactValid[i]),
                                    mySphere,
                                    sharedSpheres);
		__syncthreads();
    }

    // Save the result in global memory for the collision response step.
    for (i=0;i < numCols;i++)
    {
        //Coalesce memory accesses with float4s instead.
		globalContactNormal[i].x = currentContactNormal[i].x;
		globalContactNormal[i].y = currentContactNormal[i].y;
		globalContactNormal[i].z = currentContactNormal[i].z;
		globalContactNormal[i].w = 0;
        globalContactPointPenetration[i] = currentContactPointPenetration[i];
        globalContactValidBodies[i].x = currentContactValid[i];
		globalContactValidBodies[i].y = gtid; //Set body1 index
		globalContactValidBodies[i].z = (i+gtid+1); //Set body2 index
    }
}

__host__ void
copySpheresToFloat4s(float4* sphereFloat4s,
						jfCollisionSphereStruct* sphereStructs,
						unsigned sphereSize)
{
	for(unsigned i = 0; i < sphereSize ; i++)
	{
		sphereFloat4s[i].x = sphereStructs[i].m_Centre.m_X;
		sphereFloat4s[i].y = sphereStructs[i].m_Centre.m_Y;
		sphereFloat4s[i].z = sphereStructs[i].m_Centre.m_Z;
		sphereFloat4s[i].w = sphereStructs[i].m_Radius; //Use w value as radius.
	}
}

__host__ void
generateContacts(jfContactStruct contactStructs[N_CONTACTS],
					float4 contactNormal[N_CONTACTS],
					float4 contactPointPenetration[N_CONTACTS],
					int4 contactValid[N_CONTACTS],
					unsigned numberContacts)
{
	for(unsigned i = 0; i < numberContacts ; i++)
	{
		if(contactValid[i].x)
		{
			contactStructs[i].m_ContactNormal.m_X = contactNormal[i].x;
			contactStructs[i].m_ContactNormal.m_Y = contactNormal[i].y;
			contactStructs[i].m_ContactNormal.m_Z = contactNormal[i].z;
			contactStructs[i].m_ContactPoint.m_X = contactPointPenetration[i].x;
			contactStructs[i].m_ContactPoint.m_Y = contactPointPenetration[i].y;
			contactStructs[i].m_ContactPoint.m_Z = contactPointPenetration[i].z;
			contactStructs[i].m_Penetration = contactPointPenetration[i].w;
			contactStructs[i].m_Valid = contactValid[i].x;
			contactStructs[i].m_Body1Idx = contactValid[i].y;
			contactStructs[i].m_Body2Idx = contactValid[i].z;
		}
	}
}

/**
 * Wrapper function invoked by "jfCollisionDetector_cuda"
 * class in sphereAndSphereBatch method.
 */
extern "C" void
sphereSphereCollisionTiled(jfCollisionSphereStruct* sphereStructs,
                                jfContactStruct* contactStructs)
{
    float4 contactNormal[N_CONTACTS];
    float4 contactPointPenetration[N_CONTACTS]; //Holds point and penetration
    int4 contactValidBodies[N_CONTACTS]; //Holds validity and body pointers
	float4 sphere[N_SPHERES];

	//Copy spheres over to local float4s
	copySpheresToFloat4s(sphere, sphereStructs, N_SPHERES);

    //Alloc mem on device
    float4* d_ContactNormal;
    hipMalloc((void**)&d_ContactNormal, N_CONTACTS*sizeof(float4));

    float4* d_ContactPointPenetration;
    hipMalloc((void**)&d_ContactPointPenetration, N_CONTACTS*sizeof(float4));

    int4* d_ContactValidBodies;
    hipMalloc((void**)&d_ContactValidBodies, N_CONTACTS*sizeof(int4));

    float4* d_Sphere;
    unsigned sizeSphere = sizeof(float4) * N_SPHERES;
    hipMalloc((void**)&d_Sphere, sizeSphere);
    hipMemcpy(d_Sphere, &sphere, sizeSphere, hipMemcpyHostToDevice);

    //Configure Kernel
    dim3 threads(BLOCK_SIZE, 1);
    dim3 grid((N_SPHERES / BLOCK_SIZE),1);

	//Call Kernel
    calculateContacts<<< grid, threads >>>(d_ContactNormal,
                                            d_ContactPointPenetration,
                                            d_ContactValidBodies,
                                            d_Sphere,
                                            N_SPHERES);
    //Get results from device
    hipMemcpy(&contactNormal,
                d_ContactNormal,
                N_CONTACTS*sizeof(float4),
                hipMemcpyDeviceToHost);

    hipMemcpy(&contactPointPenetration,
                d_ContactPointPenetration,
                N_CONTACTS*sizeof(float4),
                hipMemcpyDeviceToHost);

    hipMemcpy(&contactValidBodies,
                d_ContactValidBodies,
                N_CONTACTS*sizeof(int4),
                hipMemcpyDeviceToHost);

    //Free memory
    hipFree(d_Sphere);
    hipFree(d_ContactNormal);
    hipFree(d_ContactPointPenetration);
    hipFree(d_ContactValidBodies);

	//Fill in contact objects with contacts generated by kernel call.
	generateContacts(contactStructs, contactNormal, contactPointPenetration, contactValidBodies, N_CONTACTS);
}
